#include "hip/hip_runtime.h"
#include "kernels.h"

__global__ void calculate_positions(
	index_t* edges,
	index_t* pointStarts,
	point_t* points,
	point_t* pointsNext,
	index_t pointCount,
	point_t* velocities,
	param_t params
) {
	index_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= pointCount) return; // in case the point count isn't divisible by blockSize

	point_t thisPoint = points[idx];
	real_t fX = (real_t)0.0;
	real_t fY = (real_t)0.0;

	// calculate repelling forces
	for (index_t i = 0; i < pointCount; ++i) {
		if (i == idx) continue;

		const point_t &other = points[i];
		real_t dx = thisPoint.x - other.x;
		real_t dy = thisPoint.y - other.y;
		real_t r_2 = dx*dx + dy*dy;
		r_2 = max(r_2, 0.0001);
		real_t f = params.vertexRepulsion / r_2;
		f /= sqrt(r_2);
		fX += dx * f;
		fY += dy * f;
	}

	// calculate attracting forces
	index_t neighborsStart = pointStarts[idx];
	index_t neighborsEnd = pointStarts[idx+1];
	for (index_t i = neighborsStart; i < neighborsEnd; ++i) {
		const point_t &other = points[edges[i++]];
		index_t length = edges[i];
		real_t dx = thisPoint.x - other.x;
		real_t dy = thisPoint.y - other.y;
		real_t r_2 = dx*dx + dy*dy;
		real_t f = params.edgeCompulsion * sqrt(r_2) / (real_t)length;
		fX -= dx * f;
		fY -= dy * f;
	}

	// update velocity from force
	real_t dvx = fX * params.timeQuantum / params.vertexMass;
	real_t dvy = fY * params.timeQuantum / params.vertexMass;
	point_t &v = velocities[idx];
	v.x += dvx;
	v.y += dvy;
	v.x *= params.slowdown;
	v.y *= params.slowdown;
	
	// update position
	real_t dsx = v.x * params.timeQuantum;
	real_t dsy = v.y * params.timeQuantum;
	point_t next = points[idx];
	next.x += dsx;
	next.y += dsy;
	pointsNext[idx] = next;
}

/* kernel wrapper */
void run_calculate_positions(
	index_t* edges,
	index_t* pointStarts,
	point_t* points,
	point_t* pointsNext,
	index_t pointCount,
	point_t* velocities,
	param_t params
) {
	const index_t blocksize = 128;
	const index_t gridSize = (pointCount + blocksize - 1) / blocksize;
	calculate_positions<<<gridSize, blocksize>>>(
		edges,
		pointStarts,
		points,
		pointsNext,
		pointCount,
		velocities,
		params
	);
}